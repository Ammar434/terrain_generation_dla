#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setupRandomStatesKernel(hiprandState *states, unsigned long seed, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

void launchSetupRandomStatesKernel(hiprandState *states, unsigned long seed, int size)
{
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;

    setupRandomStatesKernel<<<numBlocks, blockSize>>>(states, seed, size);
    hipDeviceSynchronize();
}